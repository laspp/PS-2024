
#include <hip/hip_runtime.h>
// nadgrajen ščepec vectorSubtract4
// računanju razlike je dodano še kvadriranje

#ifdef __cplusplus
extern "C" {
#endif

__global__ void vectorDistance2(float *c, const float *a, const float *b, int len) {
	// določimo globalni indeks elementov
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	float diff;
	// če je niti manj kot je dolžina vektorjev, morajo nekatere izračunati več razlik
	while (gid < len) {
		diff = a[gid] - b[gid];
		c[gid] = diff * diff;
		gid += gridDim.x * blockDim.x;
	}
}

#ifdef __cplusplus
}
#endif
